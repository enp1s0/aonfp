#include <iostream>
#include <chrono>
#include <random>
#include <aonfp/aonfp.hpp>
#include <aonfp/cuda_copy.hpp>

using DEVICE_T = double;
using S_EXP_T = uint32_t;
using MANTISSA_T = uint32_t;

constexpr std::size_t N = 1lu << 20;

int main() {
	std::mt19937 mt(std::random_device{}());
	std::uniform_real_distribution<DEVICE_T> dist(-1000.0, 1000.0);

	DEVICE_T* device_array;
	S_EXP_T* s_exp_array;
	MANTISSA_T* mantissa_array;

	hipMalloc(&device_array, sizeof(DEVICE_T) * N);
	hipHostMalloc(&s_exp_array, sizeof(S_EXP_T) * N);
	hipHostMalloc(&mantissa_array, sizeof(MANTISSA_T) * N);

	for (std::size_t i = 0; i < N; i++) {
		aonfp::decompose(s_exp_array[i], mantissa_array[i], dist(mt));
	}

	hipDeviceSynchronize();
	const auto start_clock = std::chrono::system_clock::now();
	aonfp::cuda::copy_to_device(device_array, s_exp_array, mantissa_array, N);
	hipDeviceSynchronize();
	const auto end_clock = std::chrono::system_clock::now();

	const auto elapsed_time = std::chrono::duration_cast<std::chrono::microseconds>(end_clock - start_clock).count() * 1.e-6;

	std::printf("Bandwidth between host and device memory : %e [GiB/s]\n", N * (sizeof(S_EXP_T) + sizeof(MANTISSA_T)) / elapsed_time / (1lu << 30));

	hipHostFree(s_exp_array);
	hipHostFree(mantissa_array);
}
